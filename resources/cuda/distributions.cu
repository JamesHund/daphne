
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cmath>

// Normal Distribution
__device__ float sample_normal(hiprandState* state, float mean, float stddev) {
    return mean + stddev * hiprand_normal(state);
}

__device__ float log_prob_normal(float x, float mean, float stddev) {
    float variance = stddev * stddev;
    return -0.5 * logf(2 * M_PI * variance) - (x - mean) * (x - mean) / (2 * variance);
}

// Laplace Distribution
__device__ float sample_laplace(hiprandState* state, float mean, float scale) {
    float u = hiprand_uniform(state) - 0.5;
    return mean - scale * copysignf(logf(1 - 2 * fabsf(u)), u);
}

__device__ float log_prob_laplace(float x, float mean, float scale) {
    return -logf(2 * scale) - fabsf(x - mean) / scale;
}

// Beta Distribution
__device__ float sample_beta(hiprandState* state, float alpha, float beta) {
    float a = alpha - 1.0f, b = beta - 1.0f;
    float A = a + b;
    float B = 1.0f / (1.0f + sqrtf(2.0f * A - 1.0f));
    float C = a + (1.0f / B);
    float L = C * logf(C) - C + lgammaf(a + 1.0f) + lgammaf(b + 1.0f) - lgammaf(A + 1.0f);
    float p, u, x, y;
    do {
        u = hiprand_uniform(state);
        x = B * u;
        y = C * x;
        p = y < 1.0f ? expf(a * logf(x) + b * logf(y) - L) : 0.0f;
    } while (hiprand_uniform(state) >= p);
    return x / (x + y);
}

__device__ float log_prob_beta(float x, float alpha, float beta) {
    if (x < 0 || x > 1) return -INFINITY;
    return (alpha - 1) * logf(x) + (beta - 1) * logf(1 - x) - lgammaf(alpha) - lgammaf(beta) + lgammaf(alpha + beta);
}

// Uniform Distribution
__device__ float sample_uniform(hiprandState* state, float lower, float upper) {
    return lower + (upper - lower) * hiprand_uniform(state);
}

__device__ float log_prob_uniform(float x, float lower, float upper) {
    if (x < lower || x > upper) return -INFINITY;
    return -logf(upper - lower);
}

// Bernoulli Distribution (Discrete)
__device__ int sample_bernoulli(hiprandState* state, float p) {
    return hiprand_uniform(state) < p ? 1 : 0;
}

__device__ float log_prob_bernoulli(int x, float p) {
    return x == 1 ? logf(p) : logf(1 - p);
}

// Log-normal Distribution
__device__ float sample_log_normal(hiprandState* state, float mean, float stddev) {
    return expf(sample_normal(state, mean, stddev));
}

__device__ float log_prob_log_normal(float x, float mean, float stddev) {
    if (x <= 0) return -INFINITY;
    float log_x = logf(x);
    return log_prob_normal(log_x, mean, stddev) - log_x;
}

// Poisson Distribution (Discrete)
__device__ int sample_poisson(hiprandState* state, float lambda) {
    int k = 0;
    float L = expf(-lambda), p = 1.0;
    do {
        ++k;
        p *= hiprand_uniform(state);
    } while (p > L);
    return k - 1;
}

__device__ float log_prob_poisson(int k, float lambda) {
    if (k < 0) return -INFINITY;
    return k * logf(lambda) - lambda - lgammaf(k + 1);
}

// Exponential Distribution
__device__ float sample_exponential(hiprandState* state, float lambda) {
    return -logf(1.0f - hiprand_uniform(state)) / lambda;
}

__device__ float log_prob_exponential(float x, float lambda) {
    if (x < 0) return -INFINITY;
    return logf(lambda) - lambda * x;
}

// Custom Sampling for Gamma (Marsaglia and Tsang’s method)
__device__ float sample_gamma(hiprandState* state, float shape, float scale) {
    if (shape < 1.0) {
        float u = hiprand_uniform(state);
        return sample_gamma(state, shape + 1.0f, scale) * powf(u, 1.0f / shape);
    }
    float d = shape - 1.0f / 3.0f;
    float c = 1.0f / sqrtf(9.0f * d);
    while (true) {
        float x = hiprand_normal(state);
        float v = 1.0f + c * x;
        if (v > 0) {
            v = v * v * v;
            float u = hiprand_uniform(state);
            if (u < 1.0f - 0.0331f * (x * x) * (x * x)) return d * v * scale;
            if (logf(u) < 0.5f * x * x + d * (1.0f - v + logf(v))) return d * v * scale;
        }
    }
}

__device__ float log_prob_gamma(float x, float shape, float scale) {
    if (x <= 0) return -INFINITY;
    return (shape - 1) * logf(x) - x / scale - shape * logf(scale) - lgammaf(shape);
}
