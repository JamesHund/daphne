#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>

using namespace std;

// Helper function to check CUDA errors
void checkCudaError(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    exit(1);
  }
}

// Kernel to initialize random states
__global__ void init_random_states(hiprandState* states, int seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &states[id]);
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <num_samples>" << std::endl;
        return 1;
    }
    int N = std::stoi(argv[1]);  // Number of samples
    int K = %d; // Number of sample statements
    int L = %d; // Number of observation statements

    // Allocate device memory for samples, observations, and results
    float* samples;
    float* log_probs;
    hiprandState* rand_states;
    checkCudaError(hipMalloc((void**)&samples, N * K * sizeof(float)));
    checkCudaError(hipMalloc((void**)&log_probs, N * L * sizeof(float))); 
    checkCudaError(hipMalloc((void**)&rand_states, N * sizeof(hiprandState)));

    // Initialize random states
    int seed = 0;
    init_random_states<<<(N + 255) / 256, 256>>>(rand_states, seed);
    hipDeviceSynchronize();

    // Launch the combined sampling and observation kernel
    generate_samples<<<(N + 255) / 256, 256>>>(rand_states, samples, log_probs, N, K);
    hipDeviceSynchronize();

    // Copy results back to host
    float* res_samples = new float[N * K];
    float* res_log_probs = new float[N * L];
    checkCudaError(hipMemcpy(res_samples, samples, N * K * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(res_log_probs, log_probs, N * L * sizeof(float), hipMemcpyDeviceToHost));

    // Print samples and log_probs into JSON
    cout << "{";
    cout << "\"samples\":[\n";
    for (int i = 0; i < N; ++i) {
        cout << "[";
        for (int j = 0; j < K; ++j) {
            cout << res_samples[i * K + j];
            if (j < K - 1) cout << ",";
        }
        cout << "]";
        if (i < N - 1) cout << ",\n";
    }
    cout << "],\n";
    cout << "\"log_probs\":[\n";
    for (int i = 0; i < N; ++i) {
        cout << "[";
        for (int j = 0; j < L; ++j) {
            cout << res_log_probs[i * L + j];
            if (j < L - 1) cout << ",";
        }
        cout << "]";
        if (i < N - 1) cout << ",\n";
    }
    cout << "]";
    cout << "}";


    // Free device memory
    checkCudaError(hipFree(samples));
    checkCudaError(hipFree(log_probs));
    checkCudaError(hipFree(rand_states));
    delete[] res_samples;
    delete[] res_log_probs;

    return 0;
}
